#include "hip/hip_runtime.h"
#include "gelu_cuda.h"
#include <cmath>
#include <hip/hip_runtime.h>
#include <>

const float sq = 0.797884f;

__global__ void geluKernel(const float* input, float* output, size_t size) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        const float x = input[idx];
        output[idx] = 0.5f * x * (1.0f + tanhf(sq * x * (1.0f + 0.044715f * x * x)));
    }
}

std::vector<float> GeluCUDA(const std::vector<float>& input) {
    const size_t size = input.size();
    size_t bytes = size * sizeof(float);

    float* d_input;
    float* d_output;

    hipMalloc(&d_input, bytes);
    hipMalloc(&d_output, bytes);

    hipMemcpy(d_input, input.data(), bytes, hipMemcpyHostToDevice);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    size_t blocksize = deviceProp.maxThreadsPerBlock;
    size_t gridsize = (size + blocksize - 1) / blocksize;

    geluKernel<<<gridsize, blocksize>>>(d_input, d_output, size);

    std::vector<float> output(size);
    hipMemcpy(output.data(), d_output, bytes, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    return output;
}
