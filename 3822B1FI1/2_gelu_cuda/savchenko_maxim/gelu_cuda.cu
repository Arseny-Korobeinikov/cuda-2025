#include "hip/hip_runtime.h"
#include "gelu_cuda.h"
#include <cmath>
#include <hip/hip_runtime.h>

__device__ float gelu(float x) {
    const float sqrt_2_over_pi = sqrtf(2.0f / M_PI);
    return 0.5f * x * (1.0f + tanhf(sqrt_2_over_pi * (x + 0.044715f * x * x * x)));
}

__global__ void geluKernel(const float* input, float* output, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        output[idx] = gelu(input[idx]);
    }
}

std::vector<float> GeluCUDA(const std::vector<float>& input) {
    int n = input.size();
    std::vector<float> output(n);

    float *d_input, *d_output;

    hipMalloc(&d_input, n * sizeof(float));
    hipMalloc(&d_output, n * sizeof(float));

    hipMemcpy(d_input, input.data(), n * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocks = (n + threadsPerBlock - 1) / threadsPerBlock;

    geluKernel<<<blocks, threadsPerBlock>>>(d_input, d_output, n);

    hipMemcpy(output.data(), d_output, n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    return output;
}
