#include "hip/hip_runtime.h"
#include "gelu_cuda.h"
#include <hip/hip_runtime.h>
#include <>
#include <cmath>
#include <iostream>

__global__ void gelu_kernel(const float* input, float* output, int size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    float x = input[idx];
    float x3 = x * x * x;
    float arg = x + 0.044715f * x3;
    float tanh_val = 0.797885f * arg;
    float gelu_result = 0.5f * x * (1.0f + tanhf(tanh_val));
    output[idx] = gelu_result;
  }
}

std::vector<float> GeluCUDA(const std::vector<float>& input) {
  const int size = input.size();
  std::vector<float> output(size);

  float* d_input;
  float* d_output;

  hipError_t err = hipMalloc((void**)&d_input, size * sizeof(float));
  if (err != hipSuccess) {
    return output;
  }

  err = hipMalloc((void**)&d_output, size * sizeof(float));
  if (err != hipSuccess) {
    hipFree(d_input);
    return output;
  }

  err = hipMemcpy(d_input, input.data(), size * sizeof(float), hipMemcpyHostToDevice);
  if (err != hipSuccess) {
    hipFree(d_input);
    hipFree(d_output);
    return output;
  }

  const int block_size = 256;
  const int num_blocks = (size + block_size - 1) / block_size;

  gelu_kernel<<<num_blocks, block_size>>>(d_input, d_output, size);

  err = hipGetLastError();
  if (err != hipSuccess) {
    hipFree(d_input);
    hipFree(d_output);
    return output;
  }

  err = hipMemcpy(output.data(), d_output, size * sizeof(float), hipMemcpyDeviceToHost);
  if (err != hipSuccess) {
    hipFree(d_input);
    hipFree(d_output);
    return output;
  }

  hipFree(d_input);
  hipFree(d_output);

  return output;
}
