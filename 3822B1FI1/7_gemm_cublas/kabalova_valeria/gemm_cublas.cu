#include "gemm_cublas.h"
#include <hip/hip_runtime.h>
#include "hipblas.h"

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
  const std::vector<float>& b,
  int n) {

  const float alpha = 1.0f;
  const float beta = 0.0f;
  std::vector<float> c(n * n);

  float* a_gpu, * b_gpu;
  hipMalloc((void**)&a_gpu, n * n * sizeof(float));
  hipMalloc((void**)&b_gpu, n * n * sizeof(float));

  float* c_gpu;
  hipMalloc((void**)&c_gpu, n * n * sizeof(float));

  hipblasSetMatrix(n, n, sizeof(float), a.data(), n, a_gpu, n);
  hipblasSetMatrix(n, n, sizeof(float), b.data(), n, b_gpu, n);

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, b_gpu, n, a_gpu, n, &beta, c_gpu, n);
 
  hipblasGetMatrix(n, n, sizeof(float), c_gpu, n, c.data(), n);
  hipblasDestroy(handle);

  hipFree(a_gpu); hipFree(b_gpu); hipFree(c_gpu);

  return c;

}