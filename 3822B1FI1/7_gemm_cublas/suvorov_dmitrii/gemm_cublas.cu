#include "gemm_cublas.h"

std::vector<float> GemmCUBLAS(const std::vector<float>& matrix_a, const std::vector<float>& matrix_b, int dimension) {
  int buffer_size = dimension * dimension * sizeof(float);
  std::vector<float> matrix_result(dimension * dimension);

  float *device_matrix_a, *device_matrix_b, *device_result, *device_result_transposed;
  hipMalloc(&device_matrix_a, buffer_size);
  hipMalloc(&device_matrix_b, buffer_size);
  hipMalloc(&device_result, buffer_size);
  hipMalloc(&device_result_transposed, buffer_size);

  hipblasHandle_t cublas_handle;
  hipblasCreate(&cublas_handle);

  hipblasSetMatrix(dimension, dimension, sizeof(float), matrix_a.data(), dimension, device_matrix_a, dimension);
  hipblasSetMatrix(dimension, dimension, sizeof(float), matrix_b.data(), dimension, device_matrix_b, dimension);

  float alpha = 1.0f;
  float beta = 0.0f;

  hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_T, dimension, dimension, dimension, &alpha, device_matrix_a, dimension, device_matrix_b, dimension, &beta, device_result, dimension);
  hipblasSgeam(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, dimension, dimension, &alpha, device_result, dimension, &beta, nullptr, dimension, device_result_transposed, dimension);

  hipblasGetMatrix(dimension, dimension, sizeof(float), device_result_transposed, dimension, matrix_result.data(), dimension);

  hipblasDestroy(cublas_handle);

  hipFree(device_matrix_a);
  hipFree(device_matrix_b);
  hipFree(device_result);
  hipFree(device_result_transposed);

  return matrix_result;
}
