#include "gemm_cublas.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>

std::vector<float> GemmCUBLAS(const std::vector<float>& matrix_a,
                               const std::vector<float>& matrix_b,
                               int dimension) {
  std::vector<float> matrix_result(dimension * dimension);
  size_t buffer_size = dimension * dimension * sizeof(float);

  float *device_matrix_a, *device_matrix_b, *device_result, *device_result_transposed;

  hipMalloc(&device_matrix_a, buffer_size);
  hipMalloc(&device_matrix_b, buffer_size);
  hipMalloc(&device_result, buffer_size);
  hipMalloc(&device_result_transposed, buffer_size);

  hipblasSetMatrix(dimension, dimension, sizeof(float),
                  matrix_a.data(), dimension,
                  device_matrix_a, dimension);
  hipblasSetMatrix(dimension, dimension, sizeof(float),
                  matrix_b.data(), dimension,
                  device_matrix_b, dimension);

  hipblasHandle_t cublas_handle;
  hipblasCreate(&cublas_handle);

  const float alpha = 1.0f;
  const float beta = 0.0f;

  hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_T,
              dimension, dimension, dimension,
              &alpha,
              device_matrix_a, dimension,
              device_matrix_b, dimension,
              &beta,
              device_result, dimension);

  hipblasSgeam(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
              dimension, dimension,
              &alpha, device_result, dimension,
              &beta, nullptr, dimension,
              device_result_transposed, dimension);

  hipblasGetMatrix(dimension, dimension, sizeof(float),
                  device_result_transposed, dimension,
                  matrix_result.data(), dimension);

  hipFree(device_matrix_a);
  hipFree(device_matrix_b);
  hipFree(device_result);
  hipFree(device_result_transposed);

  hipblasDestroy(cublas_handle);

  return matrix_result;
}
