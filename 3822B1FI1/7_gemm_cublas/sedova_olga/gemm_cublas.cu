#include "gemm_cublas.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <vector>
std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
    if (n == 0) return {};

    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    size_t size = n * n * sizeof(float);
    float *d_a = nullptr, *d_b = nullptr, *d_c = nullptr;

    CUDA_CHECK(hipMalloc(&d_a, size));
    CUDA_CHECK(hipMalloc(&d_b, size));
    CUDA_CHECK(hipMalloc(&d_c, size));

    CUDA_CHECK(hipMemcpy(d_a, a.data(), size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, b.data(), size, hipMemcpyHostToDevice));

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasStatus_t stat = hipblasSgemm(handle,
                                     HIPBLAS_OP_N, HIPBLAS_OP_N,
                                     n, n, n,
                                     &alpha,
                                     d_b, n,
                                     d_a, n,
                                     &beta,
                                     d_c, n);

    if (stat != HIPBLAS_STATUS_SUCCESS) {
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
        hipblasDestroy(handle);
        throw std::runtime_error("cuBLAS sgemm failed");
    }

    std::vector<float> c(n * n);
    CUDA_CHECK(hipMemcpy(c.data(), d_c, size, hipMemcpyDeviceToHost));

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipblasDestroy(handle);

    return c;
}