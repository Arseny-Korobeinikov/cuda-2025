#include "gemm_cublas.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>

void transpose(const std::vector<float>& in, std::vector<float>& out, int n) {
    for (int i = 0; i < n; ++i)
        for (int j = 0; j < n; ++j)
            out[j * n + i] = in[i * n + j];
}

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float* d_A;
    float* d_B;
    float* d_C;

    size_t size = n * n * sizeof(float);
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    std::vector<float> aT(n * n), bT(n * n);
    transpose(a, aT, n);
    transpose(b, bT, n);

    hipMemcpy(d_A, aT.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, bT.data(), size, hipMemcpyHostToDevice);

    const float alpha = 1.0f;
    const float beta  = 0.0f;

    // C = alpha * A * B + beta * C
    hipblasSgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                n, n, n,
                &alpha,
                d_B, n,
                d_A, n,
                &beta,
                d_C, n);

    std::vector<float> cT(n * n);
    hipMemcpy(cT.data(), d_C, size, hipMemcpyDeviceToHost);

    std::vector<float> result(n * n);
    transpose(cT, result, n);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipblasDestroy(handle);

    return result;
}
