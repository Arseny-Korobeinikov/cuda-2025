#include "gemm_cublas.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>

void transpose(const std::vector<float>& in, std::vector<float>& out, int n) {
    for (int i = 0; i < n; ++i)
        for (int j = 0; j < n; ++j)
            out[j * n + i] = in[i * n + j];
}

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float* d_A;
    float* d_B;
    float* d_C;
	float* d_CT;

    size_t size = n * n * sizeof(float);
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);
	hipMalloc(&d_CT, size);

    std::vector<float> c(n * n);

    hipblasSetMatrix(n, n, sizeof(float), a.data(), n, d_A, n);
    hipblasSetMatrix(n, n, sizeof(float), b.data(), n, d_B, n);

    const float alpha = 1.0f;
    const float beta  = 0.0f;

    // C = alpha * A * B + beta * C
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, n, n, n, &alpha, d_A, n, d_B, n, &beta, d_C, n);
    hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, &alpha, d_C, n, &beta, nullptr, n, d_CT, n);
    hipblasGetMatrix(n, n, sizeof(float), d_CT, n, c.data(), n);

    hipblasDestroy(handle);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_CT);

    return c;
}
