#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "gemm_cublas.h"

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
  const std::vector<float>& b,
  int n) {
  std::vector<float> result(n * n, 0.f);
  int size = n * n * sizeof(float);
  float* A, *B, *C,* Ct;  // Ct - transposed matrix C

  hipMalloc(&A, size);
  hipMalloc(&B, size);
  hipMalloc(&C, size);
  hipMalloc(&Ct, size);

  hipMemcpy(A, a.data(), size, hipMemcpyHostToDevice);
  hipMemcpy(B, b.data(), size, hipMemcpyHostToDevice);

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  const float alpha = 1.f;
  const float beta = 0.f;

  hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, n, n, n, &alpha, A, n, B, n, &beta, C, n);
  hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, &alpha, C, n, &beta, nullptr, n, Ct, n);

  hipblasDestroy(handle);

  hipMemcpy(result.data(), Ct, size, hipMemcpyDeviceToHost);

  hipFree(A);
  hipFree(B);
  hipFree(C);
  hipFree(Ct);

  return result;
}