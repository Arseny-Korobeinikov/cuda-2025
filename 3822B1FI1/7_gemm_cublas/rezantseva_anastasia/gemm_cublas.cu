#include "gemm_cublas.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
    std::vector<float> result(n * n, 0.f);
    const size_t data_size = n * n * sizeof(float);

    float* d_matrix_a, * d_matrix_b, * d_matrix_c, * d_matrix_ct;
    hipMalloc(&d_matrix_a, data_size);
    hipMalloc(&d_matrix_b, data_size);
    hipMalloc(&d_matrix_c, data_size);
    hipMalloc(&d_matrix_ct, data_size);

    hipMemcpy(d_matrix_a, a.data(), data_size, hipMemcpyHostToDevice);
    hipMemcpy(d_matrix_b, b.data(), data_size, hipMemcpyHostToDevice);

    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);

    const float scale_factor = 1.0f;
    const float zero_factor = 0.0f;

    hipblasSgemm(cublas_handle,
                   HIPBLAS_OP_T,
                   HIPBLAS_OP_T,
                   n, n, n,
                   &scale_factor,
                   d_matrix_a, n,
                   d_matrix_b, n,
                   &zero_factor,
                   d_matrix_c, n);


    hipblasSgeam(cublas_handle,
                HIPBLAS_OP_T,
                HIPBLAS_OP_N,
                n, n,
                &scale_factor,
                d_matrix_c, n,
                &zero_factor,
                nullptr, n,
                d_matrix_ct, n);

    hipblasDestroy(cublas_handle);

    hipMemcpy(result.data(), d_matrix_ct, data_size, hipMemcpyDeviceToHost);

    hipFree(d_matrix_a);
    hipFree(d_matrix_b);
    hipFree(d_matrix_c);
    hipFree(d_matrix_ct);

    return result;
}