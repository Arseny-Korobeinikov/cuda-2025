#include "gemm_cublas.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
    std::vector<float> output(n * n);
    size_t bytes = n * n * sizeof(float);

    float *d_a = nullptr;
    float *d_b = nullptr;
    float *d_c = nullptr;

    hipMalloc(reinterpret_cast<void**>(&d_a), bytes);
    hipMalloc(reinterpret_cast<void**>(&d_b), bytes);
    hipMalloc(reinterpret_cast<void**>(&d_c), bytes);

    hipMemcpy(d_a, a.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), bytes, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasSgemm(handle,
                HIPBLAS_OP_N, HIPBLAS_OP_N,
                n, n, n,
                &alpha,
                d_b, n,  // B
                d_a, n,  // A
                &beta,
                d_c, n); // C

    hipMemcpy(output.data(), d_c, bytes, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipblasDestroy(handle);

    return output;
}
