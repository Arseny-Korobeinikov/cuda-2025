#include "fft_cufft.h"
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <vector>

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    int N = static_cast<int>(input.size() / (2 * batch)); // length of one signal (complex)
    if (N == 0 || batch == 0) return {};

    hipfftHandle plan;
    hipfftComplex *d_data;
    size_t size = input.size() * sizeof(float);

    hipMalloc(&d_data, size);
    hipMemcpy(d_data, input.data(), size, hipMemcpyHostToDevice);

    if (hipfftPlan1d(&plan, N, HIPFFT_C2C, batch) != HIPFFT_SUCCESS) {
        hipFree(d_data);
        throw std::runtime_error("CUFFT plan creation failed");
    }

    // Forward transform
    if (hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD) != HIPFFT_SUCCESS) {
        hipfftDestroy(plan);
        hipFree(d_data);
        throw std::runtime_error("CUFFT exec forward failed");
    }

    // Inverse transform
    if (hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD) != HIPFFT_SUCCESS) {
        hipfftDestroy(plan);
        hipFree(d_data);
        throw std::runtime_error("CUFFT exec inverse failed");
    }

    // Normalize by N
    int total_complex = N * batch;
    int total_float = total_complex * 2;
    std::vector<float> output(total_float);

    hipMemcpy(output.data(), d_data, size, hipMemcpyDeviceToHost);

    for (int i = 0; i < total_float; ++i) {
        output[i] /= N;
    }

    hipfftDestroy(plan);
    hipFree(d_data);

    return output;
}
