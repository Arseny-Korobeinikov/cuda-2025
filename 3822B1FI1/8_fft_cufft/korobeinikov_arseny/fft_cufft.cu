#include "hip/hip_runtime.h"
#include "fft_cufft.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <vector>
#include <cmath>
#include <complex>
#include <iomanip>


__global__ void NormalizeFFT(hipfftComplex* data, int count, float scale) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < count) {
        data[i].x *= scale;
        data[i].y *= scale;
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batchCount) {
    int totalElems = input.size();
    int fftLength = totalElems / (2 * batchCount);
    int numComplex = totalElems / 2;

    if (totalElems % (2 * batchCount) != 0 || fftLength == 0) {
        throw std::runtime_error("Invalid input size or batch count");
    }

    hipfftComplex* d_signal;
    hipMalloc(&d_signal, numComplex * sizeof(hipfftComplex));
    hipMemcpy(d_signal, input.data(), totalElems * sizeof(float), hipMemcpyHostToDevice);

    hipfftHandle fftPlan;
    hipfftPlan1d(&fftPlan, fftLength, HIPFFT_C2C, batchCount);

    hipfftExecC2C(fftPlan, d_signal, d_signal, HIPFFT_FORWARD);

    hipfftExecC2C(fftPlan, d_signal, d_signal, HIPFFT_BACKWARD);

    float scale = 1.0f / fftLength;
    int threads = 256;
    int blocks = (numComplex + threads - 1) / threads;
    NormalizeFFT<<<blocks, threads>>>(d_signal, numComplex, scale);

    std::vector<float> output(totalElems);
    hipMemcpy(output.data(), d_signal, totalElems * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_signal);
    hipfftDestroy(fftPlan);

    return output;
}
