#include "hip/hip_runtime.h"
#include "fft_cufft.h"
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <>
#include <iostream>

__global__ void normalizeKernel(float* array, int totalSize, float normalizationFactor) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < totalSize) {
        array[index] *= normalizationFactor;
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    int totalSize = input.size();
    int signalLength = totalSize / (2 * batch);
    std::vector<float> outputData(totalSize);
    int dataSize = sizeof(hipfftComplex) * signalLength * batch;

    hipfftHandle fftPlan;
    hipfftComplex* deviceData;

    hipMalloc(&deviceData, dataSize);
    hipMemcpy(deviceData, input.data(), dataSize, hipMemcpyHostToDevice);

    hipfftPlan1d(&fftPlan, signalLength, HIPFFT_C2C, batch);
    hipfftExecC2C(fftPlan, deviceData, deviceData, HIPFFT_FORWARD);
    hipfftExecC2C(fftPlan, deviceData, deviceData, HIPFFT_BACKWARD);

    hipDeviceProp_t deviceProperties;
    hipGetDeviceProperties(&deviceProperties, 0);
    size_t block = deviceProperties.maxThreadsPerBlock;
    size_t grid = (totalSize + block - 1) / block;

    float normalizationFactor = 1.0f / signalLength;
    normalizeKernel<<<grid, block>>>(reinterpret_cast<float*>(deviceData), totalSize, normalizationFactor);

    hipMemcpy(outputData.data(), deviceData, dataSize, hipMemcpyDeviceToHost);
    hipfftDestroy(fftPlan);
    hipFree(deviceData);

    return outputData;
}
