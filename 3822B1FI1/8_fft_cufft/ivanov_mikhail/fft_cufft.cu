#include "hip/hip_runtime.h"
#include "fft_cufft.h"
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

__global__ void normalize_vector(float* _data, size_t _length, float _norm_factor) {
  size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < _length) {
    _data[idx] *= _norm_factor;
  }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
  size_t size = input.size();
  std::vector<float> result(size, 0.f);
  int n = size / (2 * batch);
  size = size * sizeof(float);

  hipfftComplex* data_device;
  hipMalloc(&data_device, sizeof(hipfftComplex) * n * batch);
  hipMemcpy(data_device, input.data(), size, hipMemcpyHostToDevice);


  hipfftHandle plan;
  hipfftPlan1d(&plan, n, HIPFFT_C2C, batch);

  hipfftExecC2C(plan, data_device, data_device, HIPFFT_FORWARD);
  hipfftExecC2C(plan, data_device, data_device, HIPFFT_BACKWARD);

  int n_th_block = 256;
  int n_bl_grid = (size + n_th_block - 1) / n_th_block;
  float norm_factor = 1.0f / static_cast<float>(n);

  normalize_vector << <n_bl_grid, n_th_block >> > (reinterpret_cast<float*>(data_device), input.size(), norm_factor);

  hipMemcpy(result.data(), data_device, size, hipMemcpyDeviceToHost);

  hipfftDestroy(plan);
  hipFree(data_device);

  return result;
}