#include "fft_cufft.h"

#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    int n = input.size() / (2 * batch);
    size_t complex_size = sizeof(hipfftComplex);
    size_t total_size = batch * n * complex_size;

    hipfftComplex* d_data;
    hipMalloc(&d_data, total_size);
    hipMemcpy(d_data, input.data(), total_size, hipMemcpyHostToDevice);

    hipfftHandle plan;
    hipfftPlan1d(&plan, n, HIPFFT_C2C, batch);

    hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);

    hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD);

    std::vector<hipfftComplex> output(batch * n);
    hipMemcpy(output.data(), d_data, total_size, hipMemcpyDeviceToHost);

    hipFree(d_data);
    hipfftDestroy(plan);

    std::vector<float> result(batch * n * 2);
    for (int i = 0; i < batch * n; ++i) {
        result[2 * i]     = output[i].x / n;
        result[2 * i + 1] = output[i].y / n;
    }

    return result;
}
