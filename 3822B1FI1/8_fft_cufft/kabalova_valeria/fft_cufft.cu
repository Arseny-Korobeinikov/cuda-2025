#include "hip/hip_runtime.h"
#include "fft_cufft.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>


__global__ void normalize(hipfftComplex* input, int complexes, float n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < complexes) {
    hipfftComplex complex = input[i];
    input[i].x = complex.x * n;
    input[i].y = complex.y * n;
  }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
  std::vector<float> result(input.size());

  const int floats = input.size();
  const int complexes = input.size() / 2;

  hipfftComplex* data;
  hipMalloc((void**)&data, complexes * sizeof(hipfftComplex));
  hipMemcpy(data, input.data(), floats * sizeof(float), hipMemcpyHostToDevice);

  hipfftHandle plan;
  hipfftPlan1d(&plan, complexes / batch, HIPFFT_C2C, batch);
  hipfftExecC2C(plan, data, data, HIPFFT_FORWARD);

  int block_size;
  int num_blocks;
  hipOccupancyMaxPotentialBlockSize(&num_blocks, &block_size, normalize, 0, 0);
  num_blocks = (floats + block_size - 1) / block_size;
  normalize << < num_blocks, block_size >> > (data, complexes, 1.0f / (complexes / batch));

  hipfftExecC2C(plan, data, data, HIPFFT_BACKWARD);
  hipfftDestroy(plan);

  hipMemcpy(result.data(), data, floats * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(data);

  return result;
}
