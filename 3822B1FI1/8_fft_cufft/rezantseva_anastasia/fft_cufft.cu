#include "hip/hip_runtime.h"
#include "fft_cufft.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define BLOCK_SIZE 256

__global__ void normalize_complex(hipfftComplex* data, int totalComplex, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < totalComplex) {
        data[idx].x /= n;
        data[idx].y /= n;
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    const int totalFloats = input.size();
    const int n = totalFloats / (2 * batch);
    const int totalComplex = totalFloats / 2;

    hipfftHandle plan;
    hipfftPlan1d(&plan, n, HIPFFT_C2C, batch);

    hipfftComplex* gpu_data;
    hipMalloc(&gpu_data, totalComplex * sizeof(hipfftComplex));
    hipMemcpy(gpu_data, input.data(), totalFloats * sizeof(float), hipMemcpyHostToDevice);

    hipfftExecC2C(plan, gpu_data, gpu_data, HIPFFT_FORWARD);
    hipfftExecC2C(plan, gpu_data, gpu_data, HIPFFT_BACKWARD);

    int blocksPerGrid = (totalComplex + BLOCK_SIZE - 1) / BLOCK_SIZE;
    normalize_complex << <blocksPerGrid, BLOCK_SIZE >> > (gpu_data, totalComplex, n);

    std::vector<float> result(totalFloats);
    hipMemcpy(result.data(), gpu_data, totalFloats * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(gpu_data);
    hipfftDestroy(plan);

    return result;
}