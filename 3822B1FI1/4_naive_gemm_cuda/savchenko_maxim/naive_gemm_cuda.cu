#include "hip/hip_runtime.h"
#include "naive_gemm_cuda.h"
#include <hip/hip_runtime.h>

__global__ void gemmKernel(const float* a, const float* b, float* c, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; // i
    int col = blockIdx.x * blockDim.x + threadIdx.x; // j

    if (row < n && col < n) {
        float sum = 0.0f;
        for (int k = 0; k < n; ++k) {
            sum += a[row * n + k] * b[k * n + col];
        }
        c[row * n + col] = sum;
    }
}

std::vector<float> NaiveGemmCUDA(const std::vector<float>& a,
                                 const std::vector<float>& b,
                                 int n) {
    size_t size = n * n * sizeof(float);
    float *d_a, *d_b, *d_c;

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, a.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((n + 15) / 16, (n + 15) / 16);

    gemmKernel<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_c, n);

    std::vector<float> c(n * n);
    hipMemcpy(c.data(), d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return c;
}
